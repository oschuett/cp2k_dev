#include "hip/hip_runtime.h"
/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2013  Urban Borstnik and the CP2K developers group
 *****************************************************************************/

#include "dbcsr_cuda.h"

extern __shared__ double cache[];

/* The following are defined in "dbcsr_cuda.h" */
/* SQUARESIZE, BLOCK, TDIM, NUMTHREADS23SQ, BLOCKSPLIT23SQ, TOTALTHREADS23SQ */

/******************************************************************************/
/* The stack_mm_mnk_sq23_d routine was added by Neil Stringfellow */
/* This is a specialised routine for square matrices of size 23 (m==n==k==23) */
/******************************************************************************/

/* The following is only valid for square matrices of size 23 (m==n==k==23) */
/* Should be called with 64 threads only (blockDim.x==64) */
/* N.B. This is checked for in the calling routine, not down here */

/* Why 64 threads I here you ask ?  What about occupancy !!!!! */
/* !!! Update - using BLOCKSPLIT23SQ we can have 128 threads !!! */
/* I'm glad you asked ... */
/* With 64 threads we can provide the device with lots of work, and with a small
   number of threads we have a better chance of having multiple blocks being
   executed concurrently so that they will overlap core floating point computation
   and global memory traffic, hiding the memory bandwidth pressure that we saw in
   the original implemenation.
   The number of thread-blocks that can be executed concurrently is probably more
   driven by the shared memory requirements per thread-block which is a little
   over 8Kbytes, and therefore we should be able to get up to 5 blocks working at
   the same time per SM.
   If indeed we can have a maximum of 5 thread-blocks each of 64 threads working
   concurrently then we have at most 320 threads, and these threads then can use
   the full complement of 63 32-bit registers available as a maximum per thread.
   This then means that we can hold up to 31 double precision numbers in registers
   and allows the inner kernel (see the code) to operate C=Matmul(A,B) on 3x3
   square matrices entirely out of registers (3 arrays each of 3x3 = 27 registers
   required for the inner kernel */

/* Needs 24x24*8*2 bytes of shared memory when being called */
__global__ void stack_mm_mnk_sq23_d (
	const int *__restrict__ param_stack,
	const int careful, const int nruns,
	const int m, const int n, const int k,
	//const int mn, const int mk, const int kn, const int maxb,
	const int liter,
	const double *__restrict__ a_data,
	const double *__restrict__ b_data,
	double *__restrict__ c_data,
	int *__restrict__ c_locks) {
	
	/**
	 *  \var sp        which stack member this thread block is processing
	 (= CUDA thread block)
	 *  \var psp       pointer to first element of parameters
	 *  \var c_loc     pointer to C data
	 *  \var run       run number
         *  \var nrun      number of runs
	 *  \var my_id     my ID for locking
	 *  \var tn        thread number (of CUDA thread block)
	 *  \var mn        product of the block dimensions
	 *  \var l         multiplication loop index
	 *  \var c, r      C matrix row, column of this thread
	 *  \var myc       C matrix accumulator
	 *  \var buff_l    cache for A data
	 *  \var buff_r    cache for B data
	 *  \var c_id      translated C block number (used in locking)
	 *  \var lock_owner  current C block owner (used in locking)
	 */ 

	int lock_owner, c_id, my_id;

	/* We have received the data from a Fortran code, so we should try to keep
	   Fortran ordering when considering rows and columns */

	/* Each thread has a row and column beginning number.
	   In each case we have an "r" value which is defined as how many rows we move down the matrix to get our
	   starting point, and this is then multiplied by blocks of (BLOCKS/TDIM) for the next entry.
	   This has changed from the original implementation in case it helps with bank conflicts.
	   For the starting column we have to move along according to TDIM, and here "c" will be
	   later multiplied by TDIM to find the starting point.
	*/
	const int r = (threadIdx.x % NUMTHREADS23SQ) % (BLOCK/TDIM);
	const int c = (threadIdx.x % NUMTHREADS23SQ) / (BLOCK/TDIM);
	const int whichhalf=threadIdx.x/(NUMTHREADS23SQ);
	const int blockbegin=((BLOCK/BLOCKSPLIT23SQ)*whichhalf)/(TDIM);
	const int a_offset_base=r;
	const int b_offset_base=TDIM*c*BLOCK;
	const int a_offset=a_offset_base + blockbegin*TDIM*BLOCK;
	const int b_offset=b_offset_base + blockbegin*TDIM;

	int l, i;

	/* We convert myc into mycarr, a TDIMxTDIM array (i.e. mycarr = myc-array) */
	double mycarr[TDIM][TDIM];

	const double * __restrict__ buff_a, * __restrict__ buff_b;
       
	int psp, c_loc;

	int run, nrun;

	double *buff;

	int j, ki;

	int myarrayindex;
	int buffaddr;

	buff = (double *) cache;

	/* We're going to use a 24x24 block of shared memory to store our 23x23 A and B matrices */
	/* This will allow us to give an independent 3x3 block to each of 64 threads in a 8x8 configuration */
	/* The edge threads will do some redundant calculations, but this is less important than
	   having a balanced workload and clean code. The actual flops are about 9% more than the flops we
	   want, but the threads would be scheduled to do something anyway, and we would need to sprinkle
	   if statements throughout the code. */
	/* So we have buff_a as an alias for the first 24x24 (x8byte double precision) block of shared
	   memory and buff_b as the second part */
	buff_a = buff;
	buff_b = &(buff[BLOCK*BLOCK]);

	nrun = GROUPING;
	if (blockIdx.x == careful)
		nrun = nruns;

	/* First let's zero the 24th row and column of the A and B shared memories */
	if(threadIdx.x<BLOCK){
	  buff[BLOCK*(BLOCK-1)+threadIdx.x]=0.0l;
	  buff[BLOCK*BLOCK + BLOCK*(BLOCK-1)+threadIdx.x]=0.0l;
	  buff[BLOCK*threadIdx.x+(BLOCK-1)]=0.0l;
	  buff[BLOCK*BLOCK + BLOCK*threadIdx.x+(BLOCK-1)]=0.0l;
	}

	/* Set the partial sums to zero. This used to be done in the inner loop, but now we will typically
	   carry the mycarr values over "run" loops as we only update the C matrix when we have run through
	   all iterations in this thread block that would be working on the same piece of C. */
	for(i=0;i<TDIM;i++){
	  for(j=0;j<TDIM;j++){
	    mycarr[i][j] = 0.0l;
	  }
	}

	for (run = 0; run < nrun; run ++) {
		psp = 7*(blockIdx.x*GROUPING + run);

		/* Load from main memory  and store into A and B */
		/* We won't get coallesced accesses on 128-byte boundaries, but we should be close to the
		   optimal performance since we are contiguous in memory and therefore the remainder of
		   each 128-byte boundary should have been loaded in L2 cache. */
		/* We need to map from the 23x23 arrays in a_data and b_data to a 24x24 block that 
		 we will use in shared memory, with the 24th row and column being zero. */

		{
		  const int a_arr_base_index=param_stack[psp+3]-1;
		  const int b_arr_base_index=param_stack[psp+4]-1;

		  for (l=0;l<(SQUARESIZE*SQUARESIZE)/TOTALTHREADS23SQ+1;l++){
		    int myarrayindex=l*TOTALTHREADS23SQ+threadIdx.x;
		    int buffaddr=myarrayindex + (myarrayindex)/SQUARESIZE;
		    if(myarrayindex<(SQUARESIZE*SQUARESIZE)){
		      /* Copy A array */
		      buff[buffaddr]=a_data[a_arr_base_index+myarrayindex];
		      /* Copy B array */
		      buff[(BLOCK*BLOCK)+buffaddr]=b_data[b_arr_base_index+myarrayindex];
		    }
		  }
		  
		}

		syncthreads();

		/* Do multiplication in 3x3 blocks over the appropriate rows/columns of A and B */
		for (l = 0; l < ((BLOCK)/((TDIM)*(BLOCKSPLIT23SQ))); l++) {
		  /* Declare two arrays for a and b that we hope the compiler will place in registers */
		  /* Actually we don't need arrays strictly in the algorithm so let's declare them a
		     little differently - looking at the assembler output with unrolling you get the same
		     effect with either configuration anyway. */
		  double a_reg_scalar;
		  double b_regs1D[TDIM];

		  /* Load the A and B values into registers and compute using those registers */
		  /* This is a rearrangement from the original just to show that the register load and
		     compute can be done together */
#pragma unroll
		  for(ki=0;ki<TDIM;ki++){
#pragma unroll
		    for (i=0;i<TDIM;i++){
		      a_reg_scalar=buff_a[   ((l*TDIM)+ki)*BLOCK  + a_offset + i*(BLOCK/TDIM) ];
#pragma unroll
		      for (j=0;j<TDIM;j++){
			if(i==0){
			  b_regs1D[j]=buff_b[   b_offset + j*BLOCK + (l*TDIM) + ki ];
			}
			  mycarr[i][j] += a_reg_scalar * b_regs1D[j];
		      }
		    }
		  }
		}

		/* Only update c_data if we are in the last iteration, or if the next C-block
		   will be different to this C-block */
		/* param_stack[psp+6] is the current C-block ID, so adding 7 means that param_stack[psp+6+7]
		   should be the next C-block ID */
		if(run==nrun-1 || param_stack[psp+6]-1 != param_stack[psp+6+7]-1) {
		  c_loc = param_stack[psp+5]-1;
		  c_id = param_stack[psp+6]-1;
		  
		  if (threadIdx.x == 0) {
		    my_id = blockIdx.x+1;
		    lock_owner = 0;
		    while ((lock_owner != my_id))
		      lock_owner = atomicCAS (&(c_locks[c_id]), 0, my_id);
		  } 
		  
		  /* Here we need to treat the threads differently depending upon whether they are 
		     in the lower half or the upper half of the thread block in the case of a 128-thread
		     thread block. All threads in the same warp will have the same value for "whichhalf" so
		     hopefully this should work smoothly. */
		  for(ki=0;ki<BLOCKSPLIT23SQ;ki++){
		  
		  /* Need to have finished with A in order to reuse shared memory used for A for a
		     temporary store, so we need a sync */
		    syncthreads();
		    
		    if(ki==whichhalf){
		      
		      /* Add our results into a temporary storage in buff that is normally used for A.
			 This forms our C block in the same 24x24 form as for A and B in shared memory.
			 As we need to refresh A on every run through the "run" loop, we can reuse buff
			 for storing this copy of C. */
		      if(ki==0){
#pragma unroll
			for (i=0;i<TDIM;i++){
#pragma unroll
			  for (j=0;j<TDIM;j++){
			    buff[a_offset_base + b_offset_base + j*BLOCK + i*(BLOCK/TDIM) ]=mycarr[i][j];
			  }
			}
			
		      }else{
			
#pragma unroll
			for (i=0;i<TDIM;i++){
#pragma unroll
			  for (j=0;j<TDIM;j++){
			    buff[a_offset_base + b_offset_base + j*BLOCK + i*(BLOCK/TDIM) ]+=mycarr[i][j];
			  }
			}
			
		      }
		    }
		  }

		  /* We need to ensure that we have a coherent copy of C in the buffer, so that
		     means we have another sync */
		  syncthreads();

		  /* Need to reverse engineer back where these go into memory */
		  /* Need to effect coallesced accesses so as when loading A and B earlier, we use
		   a mapping of the 23x23 C block onto our 24x24 buff block where row 24 and column
		   24 are zeroes so they can be ignored. */
		  for (l=0;l<=(SQUARESIZE*SQUARESIZE)/TOTALTHREADS23SQ+1;l++){
		    myarrayindex=l*TOTALTHREADS23SQ+threadIdx.x;
		    if(myarrayindex<(SQUARESIZE*SQUARESIZE)){
		      buffaddr=myarrayindex + (myarrayindex)/SQUARESIZE;
		      c_data[c_loc+myarrayindex]+=buff[buffaddr];
		    }
		  }

		  /* Release the lock on the C block. */
		  syncthreads();
		  if (threadIdx.x == 0) {
		    c_locks[c_id] = 0;
		  }
		  /* If we have another C-block then we need to reset our partial sum to zero for the new C-block */
		  if(run!=nrun-1){
		    for(i=0;i<TDIM;i++){
		      for(j=0;j<TDIM;j++){
			mycarr[i][j] = 0.0l;
		      }
		    }
		  }

		}

		syncthreads();

	}

};

__global__ void stack_mm_mnk_d (
	const int *__restrict__ param_stack,
	const int careful, const int nruns,
	const int m, const int n, const int k,
	//const int mn, const int mk, const int kn, const int maxb,
	const int liter,
	const double *__restrict__ a_data,
	const double *__restrict__ b_data,
	double *__restrict__ c_data,
	int *__restrict__ c_locks) {
	
	/**
	 *  \var sp        which stack member this thread block is processing
	 (= CUDA thread block)
	 *  \var psp       pointer to first element of parameters
	 *  \var c_loc     pointer to C data
	 *  \var run       run number
         *  \var nrun      number of runs
	 *  \var my_id     my ID for locking
	 *  \var tn        thread number (of CUDA thread block)
	 *  \var mn        product of the block dimensions
	 *  \var l         multiplication loop index
	 *  \var c, r      C matrix row, column of this thread
	 *  \var myc       C matrix accumulator
	 *  \var buff_l    cache for A data
	 *  \var buff_r    cache for B data
	 *  \var c_id      translated C block number (used in locking)
	 *  \var lock_owner  current C block owner (used in locking)
	 */ 

	int lock_owner, c_id, my_id;
	const int mn = m * n;
	const int mk = m * k;
	const int kn = n * k;
	const int r = threadIdx.x % m;
	const int c = threadIdx.x / m;
	int l, i;
	double myc;
	const double * __restrict__ buff_l, * __restrict__ buff_r;
       
	int psp, c_loc;

	int run, nrun;

	double *buff;

	buff = (double *) cache;
	buff_l = buff;
	buff_r = &(buff[mk]);

	nrun = GROUPING;
	if (blockIdx.x == careful)
		nrun = nruns;

	/* Set the partial sum to zero (this used to be done in the inner loop, but now we might carry it over loops) */
	myc = 0.0l;

	for (run = 0; run < nrun; run ++) {
		psp = 7*(blockIdx.x*GROUPING + run);

		for (l = 0; l <= liter; l++) {
			i = threadIdx.x+blockDim.x*l;
			if (i < mk)
				buff[i] = a_data[param_stack[psp+3]-1+i];
			if (i < kn)
				buff[mk+i] = b_data[param_stack[psp+4]-1+i];
		}

		syncthreads();

		/* Do actual multiplication. */
		if (threadIdx.x < mn) {

			for (l = 0; l < k; l++) {
				myc = myc +
				  buff_l[   l*m  + r] *
				  buff_r[   c*k + l ];
			}

		}

		/* Only update c_date if we are in the last iteration, or if the next C-block will be
		   different to this C-block */
		/* param_stack[psp+6] is the current C-block ID, so adding 7 means that param_stack[psp+6+7]
		   should be the next C-block ID */
		if(run==nrun-1 || param_stack[psp+6]-1 != param_stack[psp+6+7]-1) {
		  c_loc = param_stack[psp+5]-1;
		  c_id = param_stack[psp+6]-1;
		  
		  if (threadIdx.x == 0) {
		    my_id = blockIdx.x+1;
		    lock_owner = 0;
		    while ((lock_owner != my_id))
		      lock_owner = atomicCAS (&(c_locks[c_id]), 0, my_id);
		  } 
		  
		  
		  
		  
		  /* Add our results to the C block. */
		  syncthreads();
		  if (threadIdx.x < mn) {
		    c_data[c_loc+threadIdx.x] += myc;
		  }
		  
		  /* Release the lock on the C block. */
		  syncthreads();
		  if (threadIdx.x == 0) {
		    c_locks[c_id] = 0;
		  }
		  /* If we have another C-block then we need to reset our partial sum to zero for the new C-block */
		  myc = 0.0l;
		}

		syncthreads();

	}


};


__global__ void stack_mm_d
                   (const int *__restrict__ param_stack,
		    int stack_size, int nparams,
		    const double *__restrict__ a_data,
		    const double *__restrict__ b_data,
		    double *__restrict__ c_data,
		    int *__restrict__ c_locks) {

  /**
   *  \var sp        which stack member this thread block is processing
                     (= CUDA thread block)
   *  \var sp_one    translated stack (=sp+1)
   *  \var tn        thread number (of CUDA thread block)
   *  \var nt        number of threads (size of CUDA thread block)
   *  \var m, n, k   dimensions of the blocks (C is m*n, A is m*k, B is k*n)
   *  \var mn, mk, kn  product of the block dimensions
   *  \var l         multiplication loop index
   *  \var c, r      C matrix row, column of this thread
   *  \var myc       C matrix accumulator
   *  \var buff      cache for A and B data
   *  \var c_id      translated C block number (used in locking)
   *  \var lock_owner  current C block owner (used in locking)
   */ 

  int sp, lock_owner, c_id, sp_one;
  int tn;
  int r, c, l;
  int m, n, k;
  int mn;
  double myc;
  const double *buff_l, *buff_r;

  int psp, c_loc;


  /* Setup shared memory. */
  //buff = (double *) cache;

  /* Determine who I am. */
  sp = blockIdx.x;
  tn = threadIdx.x;

  psp = 7*sp;
  m = param_stack[psp];
  n = param_stack[psp+1];
  k = param_stack[psp+2];

  buff_l = &(a_data[param_stack[psp+3]-1]);
  buff_r = &(b_data[param_stack[psp+4]-1]);

  /* Calculate who I am. */

  mn = m*n;

  /* Do actual multiplication. */
  if (tn < mn) {
    r = tn % m;
    c = tn / m;
    myc = 0.0l;

    for (l = 0; l < k; l++) {
      myc = myc +
	buff_l[   l*m+r] *
	buff_r[   c*k+l];
    }
  }

  /* Lock the C block. */
  c_id = param_stack[psp+6]-1;
  c_loc = param_stack[psp+5]-1;
  syncthreads();
  if (tn == 0) {
    sp_one = sp + 1;
    lock_owner = 0;
    while ((lock_owner != sp_one))
      lock_owner = atomicCAS (&(c_locks[c_id]), 0, sp_one);
  }

  /* Add our results to the C block. */
  syncthreads();
  if (tn < mn) {
    c_data[c_loc+tn] += myc;
  }

  /* Release the lock on the C block. */
  syncthreads();
  if (tn == 0) {
    c_locks[c_id] = 0;
    //threadfence();
  }

};


__global__ void stack_mm_mnk_d_direct (
	const int *__restrict__ param_stack,
	const int careful, const int nruns,
	const int m, const int n, const int k, const int mn,
	const double *__restrict__ a_data,
	const double *__restrict__ b_data,
	double *__restrict__ c_data,
	int *__restrict__ c_locks) {

	/**
	 *  \var sp        which stack member this thread block is processing
	 (= CUDA thread block)
	 *  \var psp       pointer to first element of parameters
	 *  \var c_loc     pointer to C data
	 *  \var run       run number
         *  \var nrun      number of runs
	 *  \var my_id     my ID for locking
	 *  \var tn        thread number (of CUDA thread block)
	 *  \var mn        product of the block dimensions
	 *  \var l         multiplication loop index
	 *  \var c, r      C matrix row, column of this thread
	 *  \var myc       C matrix accumulator
	 *  \var buff_l    cache for A data
	 *  \var buff_r    cache for B data
	 *  \var c_id      translated C block number (used in locking)
	 *  \var lock_owner  current C block owner (used in locking)
	 */ 

	int lock_owner, c_id, my_id;
	int l;
	const int r = threadIdx.x % m;
	const int c = threadIdx.x / m;
	double myc ;
	const double *buff_l, *buff_r;
        
	int psp, c_loc;

	int run, nrun;

	nrun = GROUPING;
	if (blockIdx.x == careful)
		nrun = nruns;

	for (run = 0; run < nrun; run ++) {
		psp = 7*(blockIdx.x*GROUPING + run);

		buff_l = &(a_data[param_stack[psp+3]-1]);
		buff_r = &(b_data[param_stack[psp+4]-1]);
		/* Do actual multiplication. */
		if (threadIdx.x < mn) {
			myc = 0.0l;

			for (l = 0; l < k; l++) {
				myc = myc +
					buff_l[   l*m+r] *
					buff_r[   c*k+l];
			}
		}

		/* Lock the C block. */
		c_loc = param_stack[psp+5]-1;
		syncthreads();
		c_id = param_stack[psp+6]-1;

		if (threadIdx.x == 0) {
			my_id = blockIdx.x+1;
			lock_owner = 0;
			while ((lock_owner != my_id))
				lock_owner = atomicCAS (&(c_locks[c_id]), 0, my_id);
		}
		

			

		/* Add our results to the C block. */
		syncthreads();
		if (threadIdx.x < mn) {
			c_data[c_loc+threadIdx.x] += myc;
		}

		/* Release the lock on the C block. */
		syncthreads();
		if (threadIdx.x == 0) {
			c_locks[c_id] = 0;
			//threadfence();
		}
	}


};


__global__ void stack_mm_mnk_vec_d (
	const int *__restrict__ param_stack,
	const int stack_size, const int nmat,
	const int m, const int n, const int k, const int mn,
	const double *__restrict__ a_data,
	const double *__restrict__ b_data,
	double *__restrict__ c_data,
	int *__restrict__ c_locks) {
	
	/**
	 *  \var sp        which stack member this thread block is processing
	 (= CUDA thread block)
	 *  \var psp       pointer to first element of parameters
	 *  \var c_loc     pointer to C data
	 *  \var run       run number
         *  \var nrun      number of runs
	 *  \var my_id    translated stack (=sp+1)
	 *  \var tn        thread number (of CUDA thread block)
	 *  \var mn        product of the block dimensions
	 *  \var l         multiplication loop index
	 *  \var c, r      C matrix row, column of this thread
	 *  \var myc       C matrix accumulator
	 *  \var buff_l    cache for A data
	 *  \var buff_r    cache for B data
	 *  \var c_id      translated C block number (used in locking)
	 *  \var lock_owner  current C block owner (used in locking)
	 */ 

	int lock_owner, c_id, my_id;
	const int tn = threadIdx.x;
	int nmat_used;
	int nt;
	const int r = threadIdx.x % m;
	int c, l;
	double myc[32];
	double mya[32];
	__shared__ int our_b[32];
	const double *buff_l, *buff_r;

	int psp, c_loc;
//	int run, nrun;
	const int my_mat_num = threadIdx.x / m;
	int imat;

	//nrun = GROUPING;
	//if ((blockIdx.x+1) * GROUPING > stack_size)
	//	nrun = stack_size - (blockIdx.x)*GROUPING;

	nmat_used = nmat;
	if ((blockIdx.x+1)*nmat > stack_size)
		nmat_used = stack_size - (blockIdx.x)*nmat;
	nt = m * nmat_used;

	//for (run = 0; run < nrun; run ++) {
	//sp = blockIdx.x*GROUPING + run;

	psp = 7*(blockIdx.x*nmat + my_mat_num);

	buff_l = &(a_data[param_stack[psp+3]-1]);
	buff_r = &(b_data[param_stack[psp+4]-1]);

	/* Do actual multiplication. */
	if (tn < nt) {
		for (l = 0; l < k; l++) {
			mya[l] = buff_l[ l*m + r ];
		}
		for (c = 0; c < n; c++) {
			if (tn < k)
				our_b[l] = buff_r[c*k+tn];
			syncthreads();
			myc[c] = 0.0l;
		
			for (l = 0; l < k; l++) {
				myc[c] = myc[c] +
					mya   [   l    ] *
					our_b [   l    ];
				//buff_r[   c*k+l];
			}
		}
	}
	/* Lock the C block. */
	c_id = param_stack[psp+6]-1;
	syncthreads();
	c_loc = param_stack[psp+5]-1;
	my_id = blockIdx.x + 1;
	for (imat = 0; imat < nmat_used; imat++) {
		if (r == 0 && imat == my_mat_num) {
			lock_owner = 0;
			while ((lock_owner != my_id))
				lock_owner = atomicCAS (&(c_locks[c_id]), 0, my_id);
		}

		/* Add our results to the C block. */
		syncthreads();
		if (tn < nt && imat == my_mat_num) {
			for (c = 0; c < n; c++) {
				c_data[c_loc+r+c*m] += myc[c];
			}
		}

		/* Release the lock on the C block. */
		syncthreads();
		if (r == 0 && imat == my_mat_num) {
			c_locks[c_id] = 0;
		}
	}
};
