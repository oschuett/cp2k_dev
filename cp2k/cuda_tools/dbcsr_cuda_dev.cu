/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2013  Urban Borstnik and the CP2K developers group
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sm_11_atomic_functions.h>

#include "dbcsr_cuda.h"

//static const int verbose_print = 0;


extern "C" int dc_device_sync_cu() {
	hipError_t cErr;
	
	cErr = hipDeviceSynchronize ();
	if (cuda_error_check (cErr)) return 1;
	return 0;
}

extern "C" int dc_stream_sync_cu(int stream_id) {
	hipError_t cErr;
	hipStream_t stream;
	
	stream = (hipStream_t) dc_get_stream (stream_id);
	cErr = hipStreamSynchronize (stream);
	if (cuda_error_check (cErr)) return 1;
	return 0;
}


extern "C" int dc_set_device_cu(int device_id) {
	hipError_t cErr;

	cErr = hipSetDevice(device_id);
	if (cuda_error_check (cErr)) return 1;
	return 0;
}

extern "C" int dc_get_ndevices_cu(int *n_devices) {
	hipError_t cErr;

	cErr = hipGetDeviceCount(n_devices);
	if (cuda_error_check (cErr)) return 1;
	return 0;
}

hipStream_t dc_get_stream (int stream_id) {
	if (nStreams == 0)
		return (hipStream_t) 0;
	else
		return streams[stream_id];
}


extern "C" int dc_create_streams(int n_streams) {
	hipError_t cErr;
	int i;

	nStreams = n_streams;
	streams = (hipStream_t *) malloc (sizeof(hipStream_t) * (n_streams+1));
	if (streams == NULL) return 2;
	for (i = 1; i <= n_streams; i++) {
		cErr = hipStreamCreate(&(streams[i]));
		if (cuda_error_check (cErr)) {
			free ((void *) streams);
			return 1;
		}
	}
	streams[0] = (hipStream_t) 0;
	return 0;
}

extern "C" int dc_destroy_streams() {
	hipError_t cErr;
	int i;

	for (i = 1; i <= nStreams; i++) {
		cErr = hipStreamDestroy(streams[i]);
		if (cuda_error_check (cErr)) {
			free ((void *) streams);
			return 1;
		}
	}

	free ((void *) streams);
	return 0;
}
