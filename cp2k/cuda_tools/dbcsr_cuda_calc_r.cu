#include "hip/hip_runtime.h"
/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2011  Urban Borstnik and the CP2K developers group
 *****************************************************************************/


__global__ void stack_mm_r
                   (const int* __restrict__ param_stack,
		    int stack_size, int nparams,
		    const float* __restrict__ a_data,
		    const float* __restrict__ b_data,
		    float* __restrict__ c_data,
		    int* __restrict__ c_locks) {

  /**
   *  \var sp        which stack member this thread block is processing
   *  \var our_params  cache for this thread block's multiplication parameters
   *  \var m, n, k   dimensions of the blocks (C is m*n, A is m*k, B is k*n)
   *  \var mn, mk, kn  product of the block dimensions
   *  \var l         multiplication loop index
   *  \var cr        C matrix element (fortran-style) of this thread
   *  \var c, r      C matrix row, column of this thread
   *  \var myc       C matrix accumulator
   *  \var buff      cache for A and B data
   */ 

  int sp, lock_owner, c_id, sp_one;
  int r, c, l;
  int m, n, k;
  int mn, cr, mk, kn;
  float myc;
  __shared__ int our_params[7];
  float *buff;


  /* Setup shared memory. */
  buff = (float *) cache;

  /* Determine who I am. */
  sp = blockIdx.x;
  cr = threadIdx.x;

  /* Load in the parameters. */
  if (blockDim.x >= 7) {
    if (cr < 7) {
      our_params[cr] = param_stack[cr+7*sp];
    }
  } else if (cr == 0) {
    for (l = 0; l < 7; l++) {
      our_params[l] = param_stack[l+7*sp];
    }
  }
  syncthreads();
  m = our_params[0];
  n = our_params[1];
  k = our_params[2];

  /* Load in the buffers.  The first mk threads load in A while the
     last kn threads load in B. */
  mk = m*k;
  kn = k*n;
  if (cr < mk)
    buff[cr] = a_data[our_params[3]-1+cr];
  if (cr >= blockDim.x - kn)
    buff[mk+cr-(blockDim.x-kn)] = b_data[our_params[4]-1+(cr-(blockDim.x-kn))];


  /* Calculate who I am. */
  syncthreads();

  mn = m*n;

  /* Do actual multiplication. */
  if (cr < mn) {
    r = cr % m;
    c = cr / m;
    myc = 0.0f;

    for (l = 0; l < k; l++) {
      myc = myc +
	buff[   l*m+r] *
	buff[mk+c*k+l];
    }
  }

  /* Lock the C block. */
  syncthreads();
  if (cr == 0) {
    sp_one = sp + 1;
    c_id = our_params[6]-1;
    lock_owner = 0;
    while ((lock_owner != sp_one))
      lock_owner = atomicCAS (&(c_locks[c_id]), 0, sp_one);
  }

  /* Add our results to the C block. */
  syncthreads();
  if (cr < mn) {
    c_data[our_params[5]-1+cr] += myc;
  }

  /* Release the lock on the C block. */
  syncthreads();
  if (cr == 0)
    c_locks[c_id] = 0;

};
